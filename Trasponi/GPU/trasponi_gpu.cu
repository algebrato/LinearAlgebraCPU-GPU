
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdio>
#include <cstdlib>
#include <time.h>
#include <math.h>

//#define DIM 16000
//#define DIM 19200
//#define DIM 22400
//#define DIM 25600
#define DIM 16392
 
#define DIM_BLOC 32
#define DEVICE 0
void inizializza(int i){

        hipDeviceProp_t prop;
        int count;

        hipGetDeviceCount (&count);

        hipGetDeviceProperties( &prop, i );
        printf("\n");
        printf("##############################################\n");
        printf("Device Name:\t\t %s\n",prop.name);
        printf("Shared Mem/block :\t %d\n",prop.sharedMemPerBlock);
        printf("Registri per blocco:\t %d\n",prop.regsPerBlock);
        printf("Warp size:\t\t %d\n",prop.warpSize);
        printf("Texature 1D :\t\t %d\n",prop.maxTexture1D);
        printf("MemPitch :\t\t %d\n",prop.memPitch);
        printf("##############################################\n");
        printf("\n");

}


__global__ void trasponi(float *out, float *idata, int larghezza, int altezza){
	__shared__ float SubMat[DIM_BLOC][DIM_BLOC];
	
	//Indice della matrice di input
	int xid = blockIdx.x * DIM_BLOC + threadIdx.x;
	int yid = blockIdx.y * DIM_BLOC + threadIdx.y;
	int indx_i = xid + yid*larghezza;
	
	//indice di quella di out ( con blocchi trasposti ) 
	int xxid = blockIdx.y * DIM_BLOC + threadIdx.x;
	int yyid = blockIdx.x * DIM_BLOC + threadIdx.y;
	int indx_o = xxid + yyid*altezza;


	for(int i=0; i<DIM_BLOC; i+=DIM_BLOC){
	       	SubMat[threadIdx.y+i][threadIdx.x] = idata[indx_i+i*larghezza];
	}

	__syncthreads();

	for(int i=0; i <DIM_BLOC; i+=DIM_BLOC){
		out[indx_o+i*altezza] = SubMat[threadIdx.x][threadIdx.y+i];
	}
	
}

__global__ void trasponiDiag(float *out, float *in, int larghezza, int altezza){
	
	__shared__ float SubMat[DIM_BLOC][DIM_BLOC+1];
	int BloccoX, BloccoY;
	
	BloccoY = blockIdx.x;
	BloccoX = (blockIdx.x+blockIdx.y)%gridDim.x;

	int xid = BloccoY * DIM_BLOC + threadIdx.x;
	int yid = BloccoX * DIM_BLOC + threadIdx.y;
	int index_i = xid +(yid)*larghezza;

	xid = BloccoY*DIM_BLOC + threadIdx.x;
	yid = BloccoX*DIM_BLOC + threadIdx.y;
	int index_o = xid + (yid)*altezza;

	for(int i=0; i<DIM_BLOC; i+=DIM_BLOC){
		SubMat[threadIdx.y+i][threadIdx.x] = in[index_i+i*larghezza];
	}
	__syncthreads();

	for(int i=0; i<DIM_BLOC; i+=DIM_BLOC){
		out[index_o+i*altezza]=SubMat[threadIdx.x][threadIdx.y+i];
	}


}



void getMatrixunitary(float* A, int NN){
	int indx=0;
	for(int i=0; i<NN; i++) for(int k=0; k<NN; k++){
		indx=k+i*NN;
		A[indx]=5.0;
	}
}



void getMatrix(float* A,int NN){
	int indx=0;
	srand(time(NULL));
	for(int i=0; i<NN; i++) for(int k=0; k<NN; k++){
		indx=k+i*NN;
	       	A[indx]=rand()%32;
	}
}

void printMatrix(float* A,int NN){
	int index=0;
	for(int i=0; i<NN; i++ ){
		for(int k=0; k<NN; k++){
			index=k+i*NN;
		       	printf("%f\t",A[index]);
		}
		printf("\n");
	}
}

int main(){
	inizializza(DEVICE);

	float* h_mat = new float [DIM*DIM];
	float* h_out = new float [DIM*DIM];
	float* d_imat = new float [DIM*DIM];
	float* d_omat = new float [DIM*DIM];
	hipEvent_t start, stop, T1,T2;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventCreate(&T1);
	hipEventCreate(&T2);
		
	dim3 block((DIM)/DIM_BLOC,(DIM)/DIM_BLOC), threads(DIM_BLOC,DIM_BLOC); 
	//riempi la matrice

	hipEventRecord(T1,0);

	getMatrix(h_mat,DIM);
	//printMatrix(h_mat,DIM);
	
	//Allocco memeoria 
	hipMalloc((void**) &d_imat, DIM*DIM*sizeof(float));
	hipMalloc((void**) &d_omat, DIM*DIM*sizeof(float));


	//copia della matrice H-->D
	hipMemcpy(d_imat,h_mat,DIM*DIM*sizeof(float), hipMemcpyHostToDevice);

	hipEventRecord(start,0);
	trasponi<<<block, threads>>>(d_omat, d_imat, DIM, DIM);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	float TempoExe;
	hipEventElapsedTime(&TempoExe,start,stop);

	printf("Tempo di Esecuzione: \t\t\t%f ms\n",TempoExe);

	hipEventRecord(start,0);
	trasponiDiag<<<block, threads>>>(d_omat, d_imat, DIM, DIM);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);

	hipEventElapsedTime(&TempoExe,start,stop);


	printf("Tempo di Esecuzione, No_CONFLICT: \t%f ms\n",TempoExe);


	hipFree(d_imat);
	hipFree(d_omat);
	free(h_mat);


	return 0;
}





